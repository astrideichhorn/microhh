#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2015 Chiel van Heerwaarden
 * Copyright (c) 2011-2015 Thijs Heus
 * Copyright (c) 2014-2015 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cstdio>
#include "grid.h"
#include "fields.h"
#include "thermo_dry.h"
#include "defines.h"
#include "constants.h"
#include "master.h"
#include "tools.h"

namespace
{
    __global__ 
    void calc_buoyancy_tend_2nd_g(double* __restrict__ wt, 
                                  double* __restrict__ th, double* __restrict__ threfh, 
                                  int istart, int jstart, int kstart,
                                  int iend,   int jend,   int kend,
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            wt[ijk] += Constants::grav/threfh[k] * (0.5*(th[ijk-kk]+th[ijk]) - threfh[k]);
        }
    }


    __global__ 
    void calc_buoyancy_g(double* __restrict__ b,
                         double* __restrict__ th, double* __restrict__ thref, 
                         int istart, int jstart,
                         int iend,   int jend,   int kcells,
                         int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z; 

        if (i < iend && j < jend && k < kcells)
        {
            const int ijk = i + j*jj + k*kk;
            b[ijk] = Constants::grav/thref[k] * (th[ijk] - thref[k]);
        }
    }

    __global__ 
    void calc_buoyancy_bot_g(double* __restrict__ b,     double* __restrict__ bbot,
                             double* __restrict__ th,    double* __restrict__ thbot, 
                             double* __restrict__ thref, double* __restrict__ threfh,
                             double grav, int kstart, int icells, int jcells,  
                             int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y; 

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            const int ijk = i + j*jj + kstart*kk;

            bbot[ij] = grav/threfh[kstart] * (thbot[ij] - threfh[kstart]);
            b[ijk]   = grav/thref [kstart] * (th[ijk]   - thref [kstart]);
        }
    }

    __global__ 
    void calc_buoyancy_flux_bot_g(double* __restrict__ bfluxbot, double* __restrict__ thfluxbot,
                                  double* __restrict__ threfh, 
                                  double grav, int kstart, int icells, int jcells,  
                                  int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y; 

        if (i < icells && j < jcells)
        {
            const int ij  = i + j*jj;
            bfluxbot[ij] = grav/threfh[kstart]*thfluxbot[ij];
        }
    }

    __global__ 
    void calc_N2_g(double* __restrict__ N2,    double* __restrict__ th,
                   double* __restrict__ thref, double* __restrict__ dzi, 
                   int istart, int jstart, int kstart,
                   int iend,   int jend,   int kend,
                   int jj, int kk)
    {
        const int i = blockIdx.x*blockDim.x + threadIdx.x + istart; 
        const int j = blockIdx.y*blockDim.y + threadIdx.y + jstart; 
        const int k = blockIdx.z + kstart; 

        if (i < iend && j < jend && k < kend)
        {
            const int ijk = i + j*jj + k*kk;
            N2[ijk] = Constants::grav/thref[k]*0.5*(th[ijk+kk] - th[ijk-kk])*dzi[k];
        }
    }
} // end namespace

void Thermo_dry::prepare_device()
{
    const int nmemsize = grid->kcells*sizeof(double);

    // Allocate fields for Boussinesq and anelastic solver
    cuda_safe_call(hipMalloc(&thref_g,   nmemsize));
    cuda_safe_call(hipMalloc(&threfh_g,  nmemsize));
    cuda_safe_call(hipMalloc(&pref_g,    nmemsize));
    cuda_safe_call(hipMalloc(&prefh_g,   nmemsize));
    cuda_safe_call(hipMalloc(&exnref_g,  nmemsize));
    cuda_safe_call(hipMalloc(&exnrefh_g, nmemsize));

    // Copy fields to device
    cuda_safe_call(hipMemcpy(thref_g,   thref,   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(threfh_g,  threfh,  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(pref_g,    pref,    nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(prefh_g,   prefh,   nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(exnref_g,  exnref,  nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(exnrefh_g, exnrefh, nmemsize, hipMemcpyHostToDevice));
    cuda_safe_call(hipMemcpy(thref_g,   thref,   nmemsize, hipMemcpyHostToDevice));
}

void Thermo_dry::clear_device()
{
    cuda_safe_call(hipFree(thref_g ));
    cuda_safe_call(hipFree(threfh_g));
    cuda_safe_call(hipFree(pref_g  ));
    cuda_safe_call(hipFree(prefh_g ));
    cuda_safe_call(hipFree(exnref_g ));
    cuda_safe_call(hipFree(exnrefh_g));
}

#ifdef USECUDA
void Thermo_dry::exec()
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kmax);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (grid->swspatialorder== "2")
    {
        calc_buoyancy_tend_2nd_g<<<gridGPU, blockGPU>>>(
            &fields->wt->data_g[offs], &fields->sp["th"]->data_g[offs], threfh_g, 
            grid->istart,  grid->jstart, grid->kstart+1,
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);

        cuda_check_error();
    }
    else if (grid->swspatialorder == "4")
    {
        master->print_message("4th order thermo_dry not (yet) implemented\n");  
        throw 1;
    }
}
#endif

#ifdef USECUDA
void Thermo_dry::get_thermo_field(Field3d *fld, Field3d *tmp, std::string name, bool cyclic)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->imax/blocki + (grid->imax%blocki > 0);
    const int gridj  = grid->jmax/blockj + (grid->jmax%blockj > 0);

    dim3 gridGPU (gridi, gridj, grid->kcells);
    dim3 blockGPU(blocki, blockj, 1);

    dim3 gridGPU2 (gridi, gridj, grid->kmax);
    dim3 blockGPU2(blocki, blockj, 1);

    const int offs = grid->memoffset;

    if (name == "b")
    {
        calc_buoyancy_g<<<gridGPU, blockGPU>>>(
            &fld->data_g[offs], &fields->sp["th"]->data_g[offs], thref_g, 
            grid->istart, grid->jstart, 
            grid->iend, grid->jend, grid->kcells,
            grid->icellsp, grid->ijcellsp);
        cuda_check_error();
    }
    else if (name == "N2")
    {
        calc_N2_g<<<gridGPU2, blockGPU2>>>(
            &fld->data_g[offs], &fields->sp["th"]->data_g[offs], thref_g, grid->dzi_g, 
            grid->istart,  grid->jstart, grid->kstart, 
            grid->iend,    grid->jend,   grid->kend,
            grid->icellsp, grid->ijcellsp);
        cuda_check_error();
    }
    else
    {
        master->print_error("get_thermo_field \"%s\" not supported\n",name.c_str());
        throw 1;
    }

    if (cyclic)
        grid->boundary_cyclic_g(&fld->data_g[offs]);
}
#endif

#ifdef USECUDA
void Thermo_dry::get_buoyancy_fluxbot(Field3d *bfield)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    calc_buoyancy_flux_bot_g<<<gridGPU, blockGPU>>>(
        &bfield->datafluxbot_g[offs], &fields->sp["th"]->datafluxbot_g[offs], 
        threfh_g, Constants::grav, grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cuda_check_error();
}
#endif

#ifdef USECUDA
void Thermo_dry::get_buoyancy_surf(Field3d *bfield)
{
    const int blocki = grid->ithread_block;
    const int blockj = grid->jthread_block;
    const int gridi  = grid->icells/blocki + (grid->icells%blocki > 0);
    const int gridj  = grid->jcells/blockj + (grid->jcells%blockj > 0);

    dim3 gridGPU (gridi, gridj, 1);
    dim3 blockGPU(blocki, blockj, 1);

    const int offs = grid->memoffset;

    calc_buoyancy_bot_g<<<gridGPU, blockGPU>>>(
        &bfield->data_g[offs], &bfield->databot_g[offs], 
        &fields->sp["th"]->data_g[offs], &fields->sp["th"]->databot_g[offs],
        thref_g, threfh_g, Constants::grav, grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cuda_check_error();

    calc_buoyancy_flux_bot_g<<<gridGPU, blockGPU>>>(
        &bfield->datafluxbot_g[offs], &fields->sp["th"]->datafluxbot_g[offs], 
        threfh_g, Constants::grav, grid->kstart, grid->icells, grid->jcells, 
        grid->icellsp, grid->ijcellsp);
    cuda_check_error();
}
#endif
